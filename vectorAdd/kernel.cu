#include "hip/hip_runtime.h"

__global__ void addOneKernel(float* out, const float* in, int numElements)
{  	
    int stride = blockDim.x * gridDim.x;
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;	
    for (; tidx < numElements; tidx += stride)
    {
        out[tidx] = in[tidx] + 1;   		
    }		
}

// Kernel Wrapper
void addOne(float* out_h, const float* in_h, int numElements)
{ 
    float *in_d, *out_d;	
    size_t size = sizeof(float) * numElements;    
    hipMalloc(&in_d, size);
    hipMalloc(&out_d, size);

    hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    addOneKernel<<<blocksPerGrid, threadsPerBlock>>>(out_d, in_d, numElements);	

    hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);

    hipFree(in_d);
    hipFree(out_d);	
}
